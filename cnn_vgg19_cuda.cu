#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "error_helper.hpp"

#define CUDA_CHECK_ERROR

#define CudaSafeCall(err) __CudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __CudaCheckError(__FILE__, __LINE__)

// global variable for counting num of threads.
__device__ unsigned long long d_totalThreads = 0;

__host__ void __CudaSafeCall(hipError_t err, const char *file, const int line) {
#ifdef CUDA_CHECK_ERROR
    if (hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line,
                hipGetErrorString(err));
        exit(-1);
    }
#endif
}

__host__ void __CudaCheckError(const char *file, const int line) {
#ifdef CUDA_CHECK_ERROR
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line,
                hipGetErrorString(err));
        exit(-1);
    }
#endif
}

// weights & bias size: (filter size * channels + 1 bias) * #filters
const float conv1_1_w = (3 * 3 * 3    + 1) * 64;
const float conv1_2_w = (3 * 3 * 64   + 1) * 64;
const float conv2_1_w = (3 * 3 * 64   + 1) * 128;
const float conv2_2_w = (3 * 3 * 128  + 1) * 128;
const float conv3_1_w = (3 * 3 * 128  + 1) * 256;
const float conv3_2_w = (3 * 3 * 256  + 1) * 256;
const float conv3_3_w = (3 * 3 * 256  + 1) * 256;
const float conv3_4_w = (3 * 3 * 256  + 1) * 256;
const float conv4_1_w = (3 * 3 * 256  + 1) * 512;
const float conv4_2_w = (3 * 3 * 512  + 1) * 512;
const float conv4_3_w = (3 * 3 * 512  + 1) * 512;
const float conv4_4_w = (3 * 3 * 512  + 1) * 512;
const float conv5_1_w = (3 * 3 * 512  + 1) * 512;
const float conv5_2_w = (3 * 3 * 512  + 1) * 512;
const float conv5_3_w = (3 * 3 * 512  + 1) * 512;
const float conv5_4_w = (3 * 3 * 512  + 1) * 512;
const float fc1_w     = (7 * 7 * 512  + 1) * 4096;
const float fc2_w     = (1 * 1 * 4096 + 1) * 4096;
const float fc3_w     = (1 * 1 * 4096 + 1) * 1000;
// layer output size
const float conv1_1  = 224 * 224 * 64;
const float conv1_2  = 224 * 224 * 64;
const float maxpool1 = 112 * 112 * 64;
const float conv2_1  = 112 * 112 * 128;
const float conv2_2  = 112 * 112 * 128;
const float maxpool2 = 56  * 56  * 128;
const float conv3_1  = 56  * 56  * 256;
const float conv3_2  = 56  * 56  * 256;
const float conv3_3  = 56  * 56  * 256;
const float conv3_4  = 56  * 56  * 256;
const float maxpool3 = 28  * 28  * 256;
const float conv4_1  = 28  * 28  * 512;
const float conv4_2  = 28  * 28  * 512;
const float conv4_3  = 28  * 28  * 512;
const float conv4_4  = 28  * 28  * 512;
const float maxpool4 = 14  * 14  * 512;
const float conv5_1  = 14  * 14  * 512;
const float conv5_2  = 14  * 14  * 512;
const float conv5_3  = 14  * 14  * 512;
const float conv5_4  = 14  * 14  * 512;
const float maxpool5 = 7   * 7   * 512;
const float fc1      = 1   * 1   * 4096;
const float fc2      = 1   * 1   * 4096;
const float fc3      = 1   * 1   * 1000;

FILE *fw;
FILE *fb;
hipblasHandle_t cubHandle;
// for cublas dummy constant
const float alpha = 1.0f;
const float beta = 0.0f;

// required to normalize by mean pixel (in rgb order)
float mean_pixel[3] = {123.68, 116.779, 103.939};
// input image
float image[224 * 224 * 3];
// ouput of each layer, device pointer
float *d_output;

__global__ void maxpooling(float *output, const float *input, const int width, const int channels)
{
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int new_width = width / 2;
    int i = thread_id / new_width * 2;
    int j = thread_id % new_width * 2;
    int index = i * width + j;

    // Atomically update num of threads.
    // XXX: Atomic operations may be relatively slow. Using such code on a regular basis is not recommended.
    atomicAdd(&d_totalThreads, 1);

    for (int c = 0; c < channels; c++) {
        float max = 0;
        if (max < input[index * channels + c])
            max = input[index * channels + c];
        if (max < input[(index + 1) * channels + c])
            max = input[(index + 1) * channels + c];
        if (max < input[(index + width) * channels + c])
            max = input[(index + width) * channels + c];
        if (max < input[(index + width + 1) * channels + c])
            max = input[(index + width + 1) * channels + c];
        output[thread_id * channels + c] = max;
    }
}

__global__ void transform_image(float *input, const float *raw_input, const int width, const int channels)
{
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int start_i = thread_id / width - 1;
    int start_j = thread_id % width - 1;
    int per_channel_width = width * width;
    int hidden_width = 3 * 3 * channels + 1;
    int global_offset = thread_id * hidden_width;

    // Atomically update num of threads.
    // XXX: Atomic operations may be relatively slow. Using such code on a regular basis is not recommended.
    atomicAdd(&d_totalThreads, 1);

    for (int c = 0; c < channels; c++) {
        int offset = 0;
        for (int i = start_i; i < start_i + 3; i++) {
            if (i < 0 || i == width)
                continue;
            for (int j = start_j; j < start_j + 3; j++) {
                if (j < 0 || j == width)
                    continue;
                input[global_offset + c * 9 + offset] = raw_input[c * per_channel_width + i * width + j];
                offset++;
            }
        }
    }
    input[(thread_id + 1) * hidden_width - 1] = 1;
}

__global__ void transform_fc(float *input, const float *raw_input, const int width, const int channels)
{
    int thread_id = threadIdx.x;
    int size = width * width;

    // Atomically update num of threads.
    // XXX: Atomic operations may be relatively slow. Using such code on a regular basis is not recommended.
    atomicAdd(&d_totalThreads, 1);

    for (int s = 0; s < size; s++)
        input[thread_id * size + s] = raw_input[s * channels + thread_id];
    if (thread_id == 0)
        input[width * width * channels] = 1;
}

__global__ void transform(float *input, const float *raw_input, const int width, const int channels)
{
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int start_i = thread_id / width - 1;
    int start_j = thread_id % width - 1;
    int hidden_width = 3 * 3 * channels + 1;
    int global_offset = thread_id * hidden_width;

    // Atomically update num of threads.
    // XXX: Atomic operations may be relatively slow. Using such code on a regular basis is not recommended.
    atomicAdd(&d_totalThreads, 1);

    float relu;
    for (int c = 0; c < channels; c++) {
        int offset = 0;
        for (int i = start_i; i < start_i + 3; i++) {
            if (i < 0 || i == width)
                continue;
            for (int j = start_j; j < start_j + 3; j++) {
                if (j < 0 || j == width)
                    continue;
                relu = raw_input[(i * width + j) * channels + c];
                input[global_offset + c * 9 + offset] = relu < 0 ? 0 : relu;
                offset++;
            }
        }
    }
    input[(thread_id + 1) * hidden_width - 1] = 1;
}

void fully_connected(int width, int channels, int num_filters)
{
    int num_weights = (width * width * channels + 1) * num_filters;
    int filter_size = width * width * channels;
    int hidden_width = filter_size + 1;
    float *weights = (float *)malloc(num_weights * sizeof(float));
    for (int i = 0; i < num_filters; i++) {
        for (int j = 0; j < filter_size; j++)
            fscanf(fw, "%f", &weights[i * hidden_width + j]);
        fscanf(fb, "%f", &weights[i * hidden_width + filter_size]);
    }

    float *d_input;
    size_t input_size = (width * width * channels + 1) * sizeof(float);
    CudaSafeCall(hipMalloc(&d_input, input_size));
    if (width == 1) {
        // previous output vector (channels * 1), expand to ((channels + 1) * 1) with a 1 at last
        float *output = (float *)malloc((channels + 1) * sizeof(float));
        CudaSafeCall(hipMemcpy(output, d_output, channels * sizeof(float), hipMemcpyDeviceToHost));
        output[channels] = 1;
        CudaSafeCall(hipMemcpy(d_input, output, (channels + 1) * sizeof(float), hipMemcpyHostToDevice));
        free(output);
    }
    else {
        // only the first fc needs to transform previous output to a vector (width * width * channels)
        transform_fc <<< 1, channels >>> (d_input, d_output, width, channels);
        CudaCheckError();
        CudaSafeCall(hipDeviceSynchronize());
    }

    float *d_weights;
    CudaSafeCall(hipMalloc(&d_weights, num_weights * sizeof(float)));
    hipFree(d_output);
    CudaSafeCall(hipMalloc(&d_output, num_filters * sizeof(float)));
    error_check(hipblasSetMatrix(hidden_width, num_filters, sizeof(float), weights, hidden_width, d_weights, hidden_width));
    // weights * input = (num_filters * (channels + 1)) * ((channels + 1) * 1), consider vector as matrix
    error_check(hipblasSgemm(cubHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, num_filters, hidden_width,
                            &alpha, d_input, 1, d_weights, hidden_width,
                            &beta, d_output, 1));

    free(weights);
    hipFree(d_input);
    hipFree(d_weights);
}

void maxpool(int width, int channels)
{
    float *d_temp;
    size_t mem_size = width * width * channels * sizeof(float);
    CudaSafeCall(hipMalloc(&d_temp, mem_size));
    CudaSafeCall(hipMemcpy(d_temp, d_output, mem_size, hipMemcpyDeviceToDevice));
    hipFree(d_output);
    CudaSafeCall(hipMalloc(&d_output, mem_size / 4));
    maxpooling <<< width / 2, width / 2 >>> (d_output, d_temp, width, channels);
    CudaCheckError();
    CudaSafeCall(hipDeviceSynchronize());
}

void convolution(int width, int channels, int num_filters)
{
    int num_weights = (3 * 3 * channels + 1) * num_filters;
    int output_size = width * width * num_filters;
    int filter_size = 3 * 3 * channels;
    int hidden_width = 3 * 3 * channels + 1;
    float *weights = (float *)malloc(num_weights * sizeof(float));
    for (int i = 0; i < num_filters; i++) {
        for (int j = 0; j < filter_size; j++)
            fscanf(fw, "%f", &weights[j * num_filters + i]);
        fscanf(fb, "%f", &weights[filter_size * num_filters + i]);
    }

    float *d_raw_input;
    float *d_input;
    size_t input_size = width * width * hidden_width * sizeof(float);
    CudaSafeCall(hipMalloc(&d_input, input_size));
    CudaSafeCall(hipMemset(d_input, 0, input_size));
    // expand original input to (width * width) * (3 * 3 * channels + 1) with a 1 at last for bias
    if (channels == 3) {
        size_t raw_input_size = width * width * channels * sizeof(float);
        CudaSafeCall(hipMalloc(&d_raw_input, raw_input_size));
        CudaSafeCall(hipMemcpy(d_raw_input, image, raw_input_size, hipMemcpyHostToDevice));
        transform_image <<< width, width >>> (d_input, d_raw_input, width, channels);
    }
    else 
        transform <<< width, width >>> (d_input, d_output, width, channels);
    CudaCheckError();
    CudaSafeCall(hipDeviceSynchronize());

    float *d_weights;
    CudaSafeCall(hipMalloc(&d_weights, num_weights * sizeof(float)));
    hipFree(d_output);
    CudaSafeCall(hipMalloc(&d_output, output_size * sizeof(float)));
    error_check(hipblasSetMatrix(num_filters, hidden_width, sizeof(float), weights, num_filters, d_weights, num_filters));
    // input * weights = ((width * width) * (3 * 3 * channels + 1)) * ((3 * 3 * channels + 1) * num_filters)
    error_check(hipblasSgemm(cubHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, num_filters, width * width, hidden_width,
                            &alpha, d_weights, num_filters, d_input, hidden_width,
                            &beta, d_output, num_filters));

    free(weights);
    if (channels == 3)
        hipFree(d_raw_input);
    hipFree(d_input);
    hipFree(d_weights);
}

// debug use, print out each element of output after a layer
void debug_print(int width, int channels, int num_filters)
{
    int output_size = width * width * channels;
    float *output = (float *)malloc(output_size * sizeof(float));
    error_check(hipblasGetMatrix(num_filters, width * width, sizeof(float), d_output, num_filters, output, num_filters));
    for (int i = 0; i < channels; i++) {
        for (int j = 0; j < width * width; j++)
            printf("%f ", output[j * channels + i]);
        printf("\n");
    }
    free(output);
}

void write_output(char *output_file)
{
    FILE *fout = fopen(output_file, "w");

    float *output = (float *)malloc(1000 * sizeof(float));
    CudaSafeCall(hipMemcpy(output, d_output, 1000 * sizeof(float), hipMemcpyDeviceToHost));

    for (int i = 0; i < 1000; i++)
        fprintf(fout, "%f\n", output[i]);

    free(output);
    hipFree(d_output);
    fclose(fout);
}

void read_image(char *image_file)
{
    FILE *fin = fopen(image_file, "r");
    int total = 224 * 224 * 3;
    for (int index = 0; index < total; index++) {
        fscanf(fin, "%f", &image[index]);
        image[index] -= mean_pixel[index / 50176]; // 50176 = 224 * 224
    }
    fclose(fin);
}

void print_device_info(void)
{
    // debug info
    int devID = 0;
    hipDeviceProp_t props;

    //Get GPU information
    CudaSafeCall(hipGetDevice(&devID));
    CudaSafeCall(hipGetDeviceProperties(&props, devID));
    printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);
}

int main(int argc, char **argv)
{
    char *image_file = argv[1];
    char *weights_file = argv[2];
    char *bias_file = argv[3];
    char *output_file = argv[4];
    unsigned long long total, tmp;
    char yellow[] = "\x1b[33m";
    char reset[] = "\x1b[39m";

    // print device info
    print_device_info();

    // read image file
    read_image(image_file);

    // initialize
    fw = fopen(weights_file, "r");
    fb = fopen(bias_file, "r");
    error_check(hipblasCreate(&cubHandle));

    // ReLU layers in transform kernel or maxpooling
    // read file input in each layer beginning to save memory cost
    convolution(224, 3, 64);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total, reset);
    tmp = total;
    convolution(224, 64, 64);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    maxpool(224, 64);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(112, 64, 128);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(112, 128, 128);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    maxpool(112, 128);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(56, 128, 256);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(56, 256, 256);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(56, 256, 256);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(56, 256, 256);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    maxpool(56, 256);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(28, 256, 512);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(28, 512, 512);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(28, 512, 512);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(28, 512, 512);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    maxpool(28, 512);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(14, 512, 512);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(14, 512, 512);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(14, 512, 512);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    convolution(14, 512, 512);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    maxpool(14, 512);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    fully_connected(7, 512, 4096); // most time consuming file input
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    fully_connected(1, 4096, 4096);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);

    tmp = total;
    fully_connected(1, 4096, 1000);
    hipMemcpyFromSymbol(&total, HIP_SYMBOL(d_totalThreads), sizeof(unsigned long long));
    printf("Total threads counted: %s%llu%s\n", yellow, total - tmp, reset);


    // write 1000 dimension
    write_output(output_file);

    fclose(fw);
    fclose(fb);
    error_check(hipblasDestroy(cubHandle));

    return 0;
}
